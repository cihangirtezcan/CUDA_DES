#include "hip/hip_runtime.h"
#include <Windows.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "tables_PRESENT.inc"
#include "tables_DES.inc"
#define Exhaustive 65536
#define THREAD 1024
#define BLOCK 512

bit64 ciphertext[THREAD * BLOCK] = { 0 };


__global__ void DES_CTR(bit64* subkey_d, bit64* ciphertext_d, bit64* expansion_table0_d, bit32* expansion_table1_d, bit64* expansion_table2_d, bit64* expansion_table3_d, bit32* s_permutation_table0_d, bit32* s_permutation_table1_d, bit32* s_permutation_table2_d, bit32* s_permutation_table3_d, bit32* s_permutation_table4_d, bit32* s_permutation_table5_d, bit32* s_permutation_table6_d, bit32* s_permutation_table7_d) {
	__shared__ bit64 expansion_table0[256], expansion_table2[256], expansion_table3[256], subkey[16];
	__shared__ bit32 expansion_table1[256], s_permutation_table0[64], s_permutation_table1[64], s_permutation_table2[64], s_permutation_table3[64], s_permutation_table4[64], s_permutation_table5[64], s_permutation_table6[64], s_permutation_table7[64];
	if (threadIdx.x < 256) {
		expansion_table0[threadIdx.x] = expansion_table0_d[threadIdx.x];
		expansion_table1[threadIdx.x] = expansion_table1_d[threadIdx.x];
		expansion_table2[threadIdx.x] = expansion_table2_d[threadIdx.x];
		expansion_table3[threadIdx.x] = expansion_table3_d[threadIdx.x];
	}
	if (threadIdx.x < 64) {
		s_permutation_table0[threadIdx.x] = s_permutation_table0_d[threadIdx.x];
		s_permutation_table1[threadIdx.x] = s_permutation_table1_d[threadIdx.x];
		s_permutation_table2[threadIdx.x] = s_permutation_table2_d[threadIdx.x];
		s_permutation_table3[threadIdx.x] = s_permutation_table3_d[threadIdx.x];
		s_permutation_table4[threadIdx.x] = s_permutation_table4_d[threadIdx.x];
		s_permutation_table5[threadIdx.x] = s_permutation_table5_d[threadIdx.x];
		s_permutation_table6[threadIdx.x] = s_permutation_table6_d[threadIdx.x];
		s_permutation_table7[threadIdx.x] = s_permutation_table7_d[threadIdx.x];
	}
	if (threadIdx.x < 16) subkey[threadIdx.x] = subkey_d[threadIdx.x];
	__syncthreads();
	bit64 temp_exp, temp;
	bit32 plaintext_left, plaintext_right;
	bit64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int i, j;
	for (i = 0; i < Exhaustive; i++) {
		plaintext_left = 0;
		plaintext_right = threadIndex + i * THREAD * BLOCK;
		for (j = 0; j < 16; j++) {
			temp_exp = expansion_table1[(plaintext_right >> 8) & 0xff] ^ expansion_table0[plaintext_right & 0xff] ^ expansion_table2[(plaintext_right >> 16) & 0xff] ^ expansion_table3[plaintext_right >> 24] ^ subkey[j];
			plaintext_left ^= s_permutation_table0[temp_exp & 0x3f] ^ s_permutation_table1[(temp_exp >> 6) & 0x3f] ^ s_permutation_table2[(temp_exp >> 12) & 0x3f] ^ s_permutation_table3[(temp_exp >> 18) & 0x3f] ^ s_permutation_table4[(temp_exp >> 24) & 0x3f] ^ s_permutation_table5[(temp_exp >> 30) & 0x3f] ^ s_permutation_table6[(temp_exp >> 36) & 0x3f] ^ s_permutation_table7[temp_exp >> 42];
			temp = plaintext_left;
			plaintext_left = plaintext_right;
			plaintext_right = temp;
		}
		temp = plaintext_left; temp = temp << 32; temp ^= plaintext_right;
		ciphertext_d[threadIndex] = temp;
	}
}
__global__ void DES_exhaustive(bit64 *plaintext_d, bit64 key2, bit32 plaintext_left2,bit32 plaintext_right2,bit32 ciphertext_left2,bit32 ciphertext_right2,bit64 *expansion_table0_d,bit32 *expansion_table1_d,bit64 *expansion_table2_d,bit64 *expansion_table3_d,bit32 *PC2_table0_d,bit32 *PC2_table1_d,bit32 *PC2_table2_d,bit64 *PC2_table3_d,bit64 *PC2_table4_d,bit64 *PC2_table5_d,bit64 *PC2_table6_d,bit32 *s_permutation_table0_d,bit32 *s_permutation_table1_d,bit32 *s_permutation_table2_d,bit32 *s_permutation_table3_d,bit32 *s_permutation_table4_d,bit32 *s_permutation_table5_d,bit32 *s_permutation_table6_d,bit32 *s_permutation_table7_d){
	__shared__ bit64 expansion_table0[256],expansion_table2[256],expansion_table3[256];
	__shared__ bit64 PC2_table3[256],PC2_table4[256],PC2_table5[256],PC2_table6[256];
	__shared__ bit32 expansion_table1[256],PC2_table0[256],PC2_table1[256],PC2_table2[256],s_permutation_table0[64],s_permutation_table1[64],s_permutation_table2[64],s_permutation_table3[64],s_permutation_table4[64],s_permutation_table5[64],s_permutation_table6[64],s_permutation_table7[64];
	if (threadIdx.x < 256) {
		expansion_table0[threadIdx.x] = expansion_table0_d[threadIdx.x];
		expansion_table1[threadIdx.x] = expansion_table1_d[threadIdx.x];
		expansion_table2[threadIdx.x] = expansion_table2_d[threadIdx.x];
		expansion_table3[threadIdx.x] = expansion_table3_d[threadIdx.x];
		PC2_table0[threadIdx.x] = PC2_table0_d[threadIdx.x];
		PC2_table1[threadIdx.x] = PC2_table1_d[threadIdx.x];
		PC2_table2[threadIdx.x] = PC2_table2_d[threadIdx.x];
		PC2_table3[threadIdx.x] = PC2_table3_d[threadIdx.x];
		PC2_table4[threadIdx.x] = PC2_table4_d[threadIdx.x];
		PC2_table5[threadIdx.x] = PC2_table5_d[threadIdx.x];
		PC2_table6[threadIdx.x] = PC2_table6_d[threadIdx.x];
	}
	if (threadIdx.x < 64) {
		s_permutation_table0[threadIdx.x] = s_permutation_table0_d[threadIdx.x];
		s_permutation_table1[threadIdx.x] = s_permutation_table1_d[threadIdx.x];
		s_permutation_table2[threadIdx.x] = s_permutation_table2_d[threadIdx.x];
		s_permutation_table3[threadIdx.x] = s_permutation_table3_d[threadIdx.x];
		s_permutation_table4[threadIdx.x] = s_permutation_table4_d[threadIdx.x];
		s_permutation_table5[threadIdx.x] = s_permutation_table5_d[threadIdx.x];
		s_permutation_table6[threadIdx.x] = s_permutation_table6_d[threadIdx.x];
		s_permutation_table7[threadIdx.x] = s_permutation_table7_d[threadIdx.x];
	}
	__syncthreads();
	bit32 plaintext_leftr=plaintext_left2, plaintext_rightr=plaintext_right2, ciphertext_left=ciphertext_left2, ciphertext_right=ciphertext_right2;
	bit64 key_real=key2+blockIdx.x*blockDim.x+threadIdx.x,temp_exp,subkey,key;
	bit32 plaintext_left,plaintext_right;
	int i,j;
	key=key_real;
	for (i=0;i<Exhaustive;i++) { 
		plaintext_left=plaintext_leftr;
		plaintext_right=plaintext_rightr;
		for (j=1;j<9;j++) {
			if (j==1 || j==5) key=((key<<1)&0xffffffeffffffe)^((key>>27)&0x10000001);
			else key=((key<<2)&0xffffffcffffffc)^((key>>26)&0x30000003);
			subkey=PC2_table0[key&0xff]^PC2_table1[(key>>8)&0xff]^PC2_table2[(key>>16)&0xff]^PC2_table3[(key>>24)&0xff]^PC2_table4[(key>>32)&0xff]^PC2_table5[(key>>40)&0xff]^PC2_table6[key>>48];
			temp_exp=expansion_table1[(plaintext_right>>8)&0xff]^expansion_table0[plaintext_right&0xff]^expansion_table2[(plaintext_right>>16)&0xff]^expansion_table3[plaintext_right>>24]^subkey;
			plaintext_left^=s_permutation_table0[temp_exp&0x3f]^s_permutation_table1[(temp_exp>>6)&0x3f]^s_permutation_table2[(temp_exp>>12)&0x3f]^s_permutation_table3[(temp_exp>>18)&0x3f]^s_permutation_table4[(temp_exp>>24)&0x3f]^s_permutation_table5[(temp_exp>>30)&0x3f]^s_permutation_table6[(temp_exp>>36)&0x3f]^s_permutation_table7[temp_exp>>42];
			if (j==1) {
				key=((key<<1)&0xffffffeffffffe)^((key>>27)&0x10000001);
				subkey=PC2_table0[key&0xff]^PC2_table1[(key>>8)&0xff]^PC2_table2[(key>>16)&0xff]^PC2_table3[(key>>24)&0xff]^PC2_table4[(key>>32)&0xff]^PC2_table5[(key>>40)&0xff]^PC2_table6[key>>48];
				temp_exp=expansion_table0[plaintext_left&0xff]^expansion_table1[(plaintext_left>>8)&0xff]^expansion_table2[(plaintext_left>>16)&0xff]^expansion_table3[plaintext_left>>24]^subkey;
				plaintext_right^=s_permutation_table0[temp_exp&0x3f]^s_permutation_table1[(temp_exp>>6)&0x3f]^s_permutation_table2[(temp_exp>>12)&0x3f]^s_permutation_table3[(temp_exp>>18)&0x3f]^s_permutation_table4[(temp_exp>>24)&0x3f]^s_permutation_table5[(temp_exp>>30)&0x3f]^s_permutation_table6[(temp_exp>>36)&0x3f]^s_permutation_table7[temp_exp>>42];

			}
			else if (j==8) {
				if (plaintext_left==ciphertext_right) {
					key=((key<<1)&0xffffffeffffffe)^((key>>27)&0x10000001);
					subkey=PC2_table0[key&0xff]^PC2_table1[(key>>8)&0xff]^PC2_table2[(key>>16)&0xff]^PC2_table3[(key>>24)&0xff]^PC2_table4[(key>>32)&0xff]^PC2_table5[(key>>40)&0xff]^PC2_table6[key>>48];
					temp_exp=expansion_table1[(plaintext_left>>8)&0xff]^expansion_table0[plaintext_left&0xff]^expansion_table2[(plaintext_left>>16)&0xff]^expansion_table3[plaintext_left>>24]^subkey;
					plaintext_right^=s_permutation_table0[temp_exp&0x3f]^s_permutation_table1[(temp_exp>>6)&0x3f]^s_permutation_table2[(temp_exp>>12)&0x3f]^s_permutation_table3[(temp_exp>>18)&0x3f]^s_permutation_table4[(temp_exp>>24)&0x3f]^s_permutation_table5[(temp_exp>>30)&0x3f]^s_permutation_table6[(temp_exp>>36)&0x3f]^s_permutation_table7[temp_exp>>42];
				}
			}
			else {
				key=((key<<2)&0xffffffcffffffc)^((key>>26)&0x30000003);
				subkey=PC2_table0[key&0xff]^PC2_table1[(key>>8)&0xff]^PC2_table2[(key>>16)&0xff]^PC2_table3[(key>>24)&0xff]^PC2_table4[(key>>32)&0xff]^PC2_table5[(key>>40)&0xff]^PC2_table6[key>>48];
				temp_exp=expansion_table1[(plaintext_left>>8)&0xff]^expansion_table0[plaintext_left&0xff]^expansion_table2[(plaintext_left>>16)&0xff]^expansion_table3[plaintext_left>>24]^subkey;
				plaintext_right^=s_permutation_table0[temp_exp&0x3f]^s_permutation_table1[(temp_exp>>6)&0x3f]^s_permutation_table2[(temp_exp>>12)&0x3f]^s_permutation_table3[(temp_exp>>18)&0x3f]^s_permutation_table4[(temp_exp>>24)&0x3f]^s_permutation_table5[(temp_exp>>30)&0x3f]^s_permutation_table6[(temp_exp>>36)&0x3f]^s_permutation_table7[temp_exp>>42];
			}
		}
		if (plaintext_right==ciphertext_left && plaintext_left==ciphertext_right) plaintext_d[0]=key_real;
		key_real+=524288; // Actually THREAD * BLOCK
		key=key_real;
	}
}
void DES_key_schedule(bit64 key, bit64 subkey[16]) {
	key = PC1_table0[key & 0xff] ^ PC1_table1[(key >> 8) & 0xff] ^ PC1_table2[(key >> 16) & 0xff] ^ PC1_table3[(key >> 24) & 0xff] ^ PC1_table4[(key >> 32) & 0xff] ^ PC1_table5[(key >> 40) & 0xff] ^ PC1_table6[(key >> 48) & 0xff] ^ PC1_table7[(key >> 56) & 0xff];
	//Round 1
	key = ((key << 1) & 0xffffffeffffffe) ^ ((key >> 27) & 0x000000010000001);
	subkey[0] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 2
	key = ((key << 1) & 0xffffffeffffffe) ^ ((key >> 27) & 0x000000010000001);
	subkey[1] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 3
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[2] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 4
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[3] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 5
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[4] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 6
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[5] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 7
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[6] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 8
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[7] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 9
	key = ((key << 1) & 0xffffffeffffffe) ^ ((key >> 27) & 0x000000010000001);
	subkey[8] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 10
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[9] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 11
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[10] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 12
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[11] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 13
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[12] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 14
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[13] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 15
	key = ((key << 2) & 0xffffffcffffffc) ^ ((key >> 26) & 0x000000030000003);
	subkey[14] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
	//Round 16
	key = ((key << 1) & 0xffffffeffffffe) ^ ((key >> 27) & 0x000000010000001);
	subkey[15] = PC2_table0[key & 0xff] ^ PC2_table1[(key >> 8) & 0xff] ^ PC2_table2[(key >> 16) & 0xff] ^ PC2_table3[(key >> 24) & 0xff] ^ PC2_table4[(key >> 32) & 0xff] ^ PC2_table5[(key >> 40) & 0xff] ^ PC2_table6[(key >> 48) & 0xff];
}
void CTR() {
	bit32* expansion_table1_d;
	bit64 key = 0x752978397493cb70, subkey[16];
	bit64* expansion_table0_d, * expansion_table2_d, * expansion_table3_d, * subkey_d;
	bit32* s_permutation_table0_d, * s_permutation_table1_d, * s_permutation_table2_d, * s_permutation_table3_d, * s_permutation_table4_d, * s_permutation_table5_d, * s_permutation_table6_d, * s_permutation_table7_d;
	bit64* ciphertext_d;
	DES_key_schedule(key, subkey);

	hipMalloc((void**)&expansion_table0_d, 256 * sizeof(bit64));	hipMemcpy(expansion_table0_d, expansion_table0, 256 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMalloc((void**)&expansion_table1_d, 256 * sizeof(bit32));	hipMemcpy(expansion_table1_d, expansion_table1, 256 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&expansion_table2_d, 256 * sizeof(bit64));	hipMemcpy(expansion_table2_d, expansion_table2, 256 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMalloc((void**)&expansion_table3_d, 256 * sizeof(bit64));	hipMemcpy(expansion_table3_d, expansion_table3, 256 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table0_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table0_d, s_permutation_table0, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table1_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table1_d, s_permutation_table1, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table2_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table2_d, s_permutation_table2, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table3_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table3_d, s_permutation_table3, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table4_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table4_d, s_permutation_table4, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table5_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table5_d, s_permutation_table5, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table6_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table6_d, s_permutation_table6, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&s_permutation_table7_d, 64 * sizeof(bit32));	hipMemcpy(s_permutation_table7_d, s_permutation_table7, 64 * sizeof(bit32), hipMemcpyHostToDevice);
	hipMalloc((void**)&subkey_d, 64 * sizeof(bit64));	hipMemcpy(subkey_d, subkey, 16 * sizeof(bit64), hipMemcpyHostToDevice);
	hipMalloc((void**)&ciphertext_d, THREAD * BLOCK * sizeof(bit64));	//hipMemset(plaintext_d, 0, THREAD * BLOCK * sizeof(bit64));
	StartCounter();
	DES_CTR << <BLOCK, THREAD >> > (subkey_d, ciphertext_d, expansion_table0_d, expansion_table1_d, expansion_table2_d, expansion_table3_d, s_permutation_table0_d, s_permutation_table1_d, s_permutation_table2_d, s_permutation_table3_d, s_permutation_table4_d, s_permutation_table5_d, s_permutation_table6_d, s_permutation_table7_d);
	hipMemcpy(ciphertext, ciphertext_d, 2 * sizeof(bit64), hipMemcpyDeviceToHost);

	printf("Time: %lf\n", GetCounter());
	printf("Ciphertext is: %I64x %I64x\n", ciphertext[0], ciphertext[1]);
//	printf("Time: %u seconds\n", clock() / CLOCKS_PER_SEC);
	// Cleanup
	hipFree(ciphertext_d); hipFree(subkey_d);
	hipFree(expansion_table0_d); hipFree(expansion_table1_d); hipFree(expansion_table2_d); hipFree(expansion_table3_d);
	hipFree(s_permutation_table0_d); hipFree(s_permutation_table1_d); hipFree(s_permutation_table2_d); hipFree(s_permutation_table3_d);
	hipFree(s_permutation_table4_d); hipFree(s_permutation_table5_d); hipFree(s_permutation_table6_d); hipFree(s_permutation_table7_d);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
}
void exhaustive() {
	bit32 *expansion_table1_d,*PC2_table0_d,*PC2_table1_d,*PC2_table2_d,plaintext_left,plaintext_right,ciphertext_left,ciphertext_right;
	bit64 plaintext=0x1122334455667788, ciphertext=0xb5219ee81aa7499d, key=0x752978397493cb70,*plaintext_d, k[2];
	bit64 *expansion_table0_d,*expansion_table2_d,*expansion_table3_d;
	bit64 *PC2_table3_d,*PC2_table4_d,*PC2_table5_d,*PC2_table6_d;
	bit32 *s_permutation_table0_d,*s_permutation_table1_d,*s_permutation_table2_d,*s_permutation_table3_d,*s_permutation_table4_d,*s_permutation_table5_d,*s_permutation_table6_d,*s_permutation_table7_d;
	key=PC1_table0[key&0xff]^PC1_table1[(key>>8)&0xff]^PC1_table2[(key>>16)&0xff]^PC1_table3[(key>>24)&0xff]^PC1_table4[(key>>32)&0xff]^PC1_table5[(key>>40)&0xff]^PC1_table6[(key>>48)&0xff]^PC1_table7[(key>>56)&0xff];
	plaintext=IP_table0[plaintext&0xff]^IP_table1[(plaintext>>8)&0xff]^IP_table2[(plaintext>>16)&0xff]^IP_table3[(plaintext>>24)&0xff]^IP_table4[(plaintext>>32)&0xff]^IP_table5[(plaintext>>40)&0xff]^IP_table6[(plaintext>>48)&0xff]^IP_table7[(plaintext>>56)&0xff];
	ciphertext=FP2_table0[ciphertext&0xff]^FP2_table1[(ciphertext>>8)&0xff]^FP2_table2[(ciphertext>>16)&0xff]^FP2_table3[(ciphertext>>24)&0xff]^FP2_table4[(ciphertext>>32)&0xff]^FP2_table5[(ciphertext>>40)&0xff]^FP2_table6[(ciphertext>>48)&0xff]^FP2_table7[(ciphertext>>56)&0xff];
	plaintext_left=plaintext>>32; plaintext_right=plaintext&0xffffffff;
	ciphertext_left=ciphertext>>32; ciphertext_right=ciphertext&0xffffffff;
	hipMalloc((void **)&expansion_table0_d, 256*sizeof(bit64));	hipMemcpy(expansion_table0_d,expansion_table0,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&expansion_table1_d, 256*sizeof(bit32));	hipMemcpy(expansion_table1_d,expansion_table1,256*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&expansion_table2_d, 256*sizeof(bit64));	hipMemcpy(expansion_table2_d,expansion_table2,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&expansion_table3_d, 256*sizeof(bit64));	hipMemcpy(expansion_table3_d,expansion_table3,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table0_d, 256*sizeof(bit32));			hipMemcpy(PC2_table0_d,PC2_table0,256*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table1_d, 256*sizeof(bit32));			hipMemcpy(PC2_table1_d,PC2_table1,256*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table2_d, 256*sizeof(bit32));			hipMemcpy(PC2_table2_d,PC2_table2,256*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table3_d, 256*sizeof(bit64));			hipMemcpy(PC2_table3_d,PC2_table3,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table4_d, 256*sizeof(bit64));			hipMemcpy(PC2_table4_d,PC2_table4,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table5_d, 256*sizeof(bit64));			hipMemcpy(PC2_table5_d,PC2_table5,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&PC2_table6_d, 256*sizeof(bit64));			hipMemcpy(PC2_table6_d,PC2_table6,256*sizeof(bit64),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table0_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table0_d,s_permutation_table0,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table1_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table1_d,s_permutation_table1,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table2_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table2_d,s_permutation_table2,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table3_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table3_d,s_permutation_table3,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table4_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table4_d,s_permutation_table4,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table5_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table5_d,s_permutation_table5,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table6_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table6_d,s_permutation_table6,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&s_permutation_table7_d, 64*sizeof(bit32));	hipMemcpy(s_permutation_table7_d,s_permutation_table7,64*sizeof(bit32),hipMemcpyHostToDevice);
	hipMalloc((void **)&plaintext_d, 2*sizeof(bit64));	hipMemset(plaintext_d,0,2*sizeof(bit64));
	StartCounter();
	DES_exhaustive<<<BLOCK, THREAD >>>(plaintext_d,key,plaintext_left,plaintext_right,ciphertext_left,ciphertext_right,expansion_table0_d,expansion_table1_d,expansion_table2_d,expansion_table3_d,PC2_table0_d,PC2_table1_d,PC2_table2_d,PC2_table3_d,PC2_table4_d,PC2_table5_d,PC2_table6_d,s_permutation_table0_d,s_permutation_table1_d,s_permutation_table2_d,s_permutation_table3_d,s_permutation_table4_d,s_permutation_table5_d,s_permutation_table6_d,s_permutation_table7_d);
	hipMemcpy(k,plaintext_d,2*sizeof(bit64),hipMemcpyDeviceToHost);
	if (k[0] || k[1]) printf("Correct key is: %I64x %I64x\n",k[1], k[0]);
	printf("Time: %lf\n", GetCounter());
	printf("Time: %u seconds\n", clock() / CLOCKS_PER_SEC);
	// Cleanup
	hipFree(plaintext_d);hipFree(expansion_table0_d);hipFree(expansion_table1_d);hipFree(expansion_table2_d);hipFree(expansion_table3_d);
	hipFree(PC2_table0_d);hipFree(PC2_table1_d);hipFree(PC2_table2_d);hipFree(PC2_table3_d);hipFree(PC2_table4_d);hipFree(PC2_table5_d);hipFree(PC2_table6_d);
	hipFree(s_permutation_table0_d);hipFree(s_permutation_table1_d);hipFree(s_permutation_table2_d);hipFree(s_permutation_table3_d);
	hipFree(s_permutation_table4_d);hipFree(s_permutation_table5_d);hipFree(s_permutation_table6_d);hipFree(s_permutation_table7_d);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
}

int main() {
	int choice = 0;
	printf(
		"(1) Exhaustive search\n"
		"(2) Counter mode\n"
		"Enter choice: "
	);
	scanf_s("%d", &choice);
	if (choice == 1) exhaustive();
	if (choice == 2) CTR();
	return 1;
}
